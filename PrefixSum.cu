#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <stdio.h>
#include<random>
using namespace std;

__global__ void prescan(float *g_odata, float *g_idata, int n)
{
	extern __shared__ float temp[];  // allocated on invocation
	int thid = threadIdx.x;
	int offset = 1;
	
	temp[2 * thid] = g_idata[2 * thid]; // load input into shared memory
	temp[2 * thid + 1] = g_idata[2 * thid + 1];
	//printf("%d - %f - %f \n", thid, g_odata[2 * thid], g_odata[2 * thid + 1]);
	//printf("%d - %f - %f \n", thid, g_idata[2 * thid], g_idata[2 * thid + 1]);
	for (int d = n >> 1; d > 0; d >>= 1)                    // build sum in place up the tree
	{
		__syncthreads();
		if (thid < d)
		{
			
			int ai = offset*(2 * thid + 1) - 1;
			int bi = offset*(2 * thid + 2) - 1;


			temp[bi] += temp[ai];
		}
		offset *= 2;
	}
	

	if (thid == 0) { temp[n - 1] = 0; } // clear the last element


	for (int d = 1; d < n; d *= 2) // traverse down tree & build scan
	{
		offset >>= 1;
		__syncthreads();
		if (thid < d)
		{
			

				int ai = offset*(2 * thid + 1) - 1;
			int bi = offset*(2 * thid + 2) - 1;


			float t = temp[ai];
			temp[ai] = temp[bi];
			temp[bi] += t;
		}
	}
	__syncthreads();
	
	g_odata[2 * thid] = temp[2 * thid]; // write results to device memory
	g_odata[2 * thid + 1] = temp[2 * thid + 1];
	
//	printf("%d - %f - %f \n", thid, g_odata[2 * thid], g_odata[2 * thid + 1]);
	//printf("%d - %f - %f \n", thid, g_idata[2 * thid], g_idata[2 * thid + 1]);
}

float * getData(int size){
	
	float *data = new float [size];
	for (int i = 0; i < size; i++){
		data[i] = std::rand()%10;
	}
	return data;
}

void displayData(float * data,int size){
	for (int i= 0; i < size; i++){
		printf("%f\n", data[i]);
	}
}
int main(){

	int size = 128;
	float *input;
	float *output = new float[size];
	float *inp_dev, *out_dev;

	input = getData(size);
//	displayData(input, size);

	hipMalloc(&inp_dev, size*sizeof(float));
	
	hipMalloc(&out_dev, size*sizeof(float));

	hipError_t cudaStatus =  hipMemcpy(inp_dev, input, size*sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	hipMemset(out_dev, 0, size*sizeof(float));
	prescan << <1, size/2, size*sizeof(float) >> >(out_dev,inp_dev, size);
	
	hipDeviceSynchronize();
	hipMemcpy(output, out_dev, size*sizeof(float), hipMemcpyDeviceToHost);

	cout << "\nFinal Output\n";
	displayData(output, size);
	
	Error:

	getchar();

	return cudaStatus;
}